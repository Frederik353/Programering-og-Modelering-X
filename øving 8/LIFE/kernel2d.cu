
#include <hip/hip_runtime.h>




__global__ void step(int *C, int *M) {
    int count;
    int n_x = blockDim.x * gridDim.x;
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    int threadId = y * n_x + x;
    int x_left; int x_right; int y_down; int y_up;

    if(x == 0) {x_left= n_x - 1;} else {x_left = x - 1;}
    if(x == n_x - 1) {x_right = 0;} else {x_right= x + 1;}
    if(y == 0) {y_down = n_x - 1;} else {y_down = y - 1;}
    if(y == n_x - 1) {y_up = 0;} else {y_up = y + 1;}


    count = C[y*n_x+x_left] + C[y_down*n_x+x]
        + C[y*n_x+x_right] + C[y_up*n_x+x] + C[y_up*n_x+x_left]
        + C[y_down*n_x+x_right] + C[y_down*n_x+x_left]
        + C[y_up*n_x+x_right];

    if(count < 2 || count > 3) M[threadId] = 0; // cell dies
    if(count == 2) M[threadId] = C[threadId];// cell stays the same
    if(count == 3) M[threadId] = 1; // cell either stays alive, or is born
}