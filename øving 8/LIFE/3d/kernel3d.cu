#include "hip/hip_runtime.h"


__global__ void step(int *C, int *M, int *neighbours, int *Neigbor_count_gpu, int *ndim) {
    int count;
    int n_x = blockDim.x*gridDim.x; // total size
    //assumes equal length in all dimensions

    int coordinates[ndim] = [threadIdx.x + blockDim.x * blockIdx.x, threadIdx.y + blockDim.y * blockIdx.y, threadIdx.z + blockDim.z * blockIdx.z];
    // int x = threadIdx.x + blockDim.x * blockIdx.x; // coordinates
    // int y = threadIdx.y + blockDim.y * blockIdx.y;
    // int z = threadIdx.z + blockDim.z * blockIdx.z;

    int threadId = y * n_x + x;

    int x_down; int x_up; 
    int y_down; int y_up;
    int z_down; int z_up;

    //lower bound
    if(x == 0) {i_down = n_x - 1;} else {i_down = x - 1;}
    if(y == 0) {y_down = n_x - 1;} else {y_down = y - 1;}
    if(z == 0) {z_down = n_x - 1;} else {z_down = z - 1;}
    // upper bound
    if(x == n_x - 1) {i_up = 0;} else {i_up = x + 1;}
    if(y == n_x - 1) {y_up = 0;} else {y_up = y + 1;}
    if(z == n_x - 1) {z_up = 0;} else {z_up = z + 1;}

    int count = 0;
    int cell_neighbours[Neigbor_count_gpu][ndim];
    for (int i = 0; i < Neigbor_count_gpu; i++){
        for (int j = 0; j < ndim; j++){
            cell_neighbours[i * ndim + j] = neighbours[i * ndim + j] * coordinates[j]
        }
        count += C[]
    }

    // count = C[j*n_x+i_left] + C[j_down*n_x+i]
    //     + C[j*n_x+i_right] + C[j_up*n_x+i] + C[j_up*n_x+i_left]
    //     + C[j_down*n_x+i_right] + C[j_down*n_x+i_left]
    //     + C[j_up*n_x+i_right];

    // Modify matrix M according to the rules B3/S23:
    //A cell is "Born" if it has exactly 3 neighbours,
    //A cell "Survives" if it has 2 or 3 living neighbours; it dies otherwise.
    if(count < 2 || count > 3) M[threadId] = 0; // cell dies
    if(count == 2) M[threadId] = C[threadId];// cell stays the same
    if(count == 3) M[threadId] = 1; // cell either stays alive, or is born
}